#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>
#include "../Image/Image.hh"
#include "../Chronometer.hh"
#include "../CPU/Morphology.hh"

__global__ void copy(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    printf("[x: %d, y: %d] ", indexX, indexY);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexY * width + indexX] = orig[indexY * width + indexX];
}

void printMat(uint8_t *mat, unsigned w, unsigned h) {

    std::cout << "[ " << std::endl;
for (int i = 0; i < w; ++i) {
        for (int j = 0; j < h; ++j) {
            std::cout << (static_cast<unsigned>(mat[i * w + j]) > 0  ? 'X' : '-') << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "]" << std::endl;
}




__global__ void erosion(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 255;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = min(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}

__global__ void dilation(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 0;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = max(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}


int ceilDivision(int a, int b) {
    if (a % b == 0)
        return a / b;
    return a / b + 1;
}


Image benchDilate(Image &img, uint8_t *kernel, int kerSide, int iterations) {
    Chronometer chrono = Chronometer("[GPU] Dilate");

    Image padded = Image::addPadding(img, kerSide / 2, 0);

    uint8_t *result = new uint8_t[img.getHeight() * img.getWidth()];

    unsigned int max = 26;
    unsigned int gridw = ceilDivision(padded.getWidth(), max);
    unsigned int gridh = ceilDivision(padded.getHeight(), max);

    dim3 grids(gridw, gridh);
    dim3 threads(max, max);


    std::cout << "padW: " << padded.getWidth() << " padH: " << padded.getHeight() << std::endl;
    std::cout << "gridw: " << gridw << " gridh: " << gridh  << " threads: " << max << std::endl;

    uint8_t *orig;
    uint8_t *morphed;
    uint8_t *ker;

    hipMalloc(&orig, sizeof (uint8_t) * (gridw * max) * (gridh * max));
    hipMalloc(&morphed, sizeof (uint8_t) * img.getWidth() * img.getHeight());
    hipMalloc(&ker, sizeof (uint8_t) * kerSide * kerSide);

    hipMemcpy(orig, padded.pixelArray(), sizeof (uint8_t) * (gridw * max) * (gridh * max), hipMemcpyHostToDevice);
    hipMemcpy(ker, kernel, sizeof (uint8_t) * kerSide * kerSide, hipMemcpyHostToDevice);

    //printMat(padded.pixelArray(), padded.getWidth(), padded.getHeight());

    dilation<<<grids, threads>>>(orig, padded.getWidth(), padded.getHeight(), morphed, img.getWidth(), ker, kerSide);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
        printf( "Error: %s\n", hipGetErrorString(err) );

    hipDeviceSynchronize();

    hipMemcpy(result, morphed, sizeof (uint8_t) * img.getWidth() * img.getHeight(), hipMemcpyDeviceToHost);

    //printMat(result, img.getWidth(), img.getHeight());

    hipDeviceSynchronize();

    hipFree(morphed);
    hipFree(orig);
    hipFree(ker);

    return Image(img.getWidth(), img.getHeight(), result);
}

int main(int argc, char **argv)
{

    Image img = Image::fromPPM("../Data/RealSnake.ppm", Image::ImportType::BINARY);
    unsigned char *kernel = Morphology::kerSquareArray(5);
    Image result = benchDilate(img, kernel, 3, 1);
    result.writePPM("result.ppm");

    return 0;
}

