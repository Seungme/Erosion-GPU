#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>

__global__ void copy(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    printf("[x: %d, y: %d] ", indexX, indexY);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexY * width + indexX] = orig[indexY * width + indexX];
}

void printMat(uint8_t *mat, unsigned w, unsigned h) {

    std::cout << "[ " << std::endl;
for (int i = 0; i < w; ++i) {
        for (int j = 0; j < h; ++j) {
            std::cout << static_cast<unsigned>(mat[i * w + j]) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "]" << std::endl;
}




__global__ void erosion(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 255;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = min(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}

__global__ void dilation(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 0;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = max(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}


int ceilDivision(int a, int b) {
    if (a % b == 0)
        return a / b;
    return a / b + 1;
}


//int main(int argc, char **argv)
//{
//
//    // img
//    uint8_t img[9] = { 1, 1, 1,
//                       1, 1, 1,
//                       1, 1, 1 };
//    unsigned int imgWidth = 13;
//    unsigned int imgHeight = 13;
//    // kernel
//    uint8_t kernel[9] = { 1, 1, 1,
//                          1, 1, 1,
//                          1, 1, 1 };
//    unsigned int kerSide = 3;
//    // paddedImg
//    /*
//    uint8_t paddedImg[25] =   { 0, 0, 0, 0, 0,
//                                0, 0, 0, 0, 0,
//                                0, 0, 1, 0, 0,
//                                0, 0, 0, 0, 0,
//                                0, 0, 0, 0, 0 };
//*/
//    uint8_t paddedImg[225] = {
//    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0,
//    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
//    };
///*
//    uint8_t *result = new uint8_t[25] { 128, 128, 128, 128, 128,
//                                        128, 128, 128, 128, 128,
//                                        128, 128, 128, 128, 128,
//                                        128, 128, 128, 128, 128,
//                                        128, 128, 128, 128, 128 };
//*/
//    uint8_t *result = new uint8_t[196]();
//
//    unsigned int paddedWidth  = imgWidth  + (kerSide  / 2) * 2;
//    unsigned int paddedHeight = imgHeight + (kerSide / 2) * 2;
//
//    unsigned int max = 8;
//    unsigned int gridw = ceilDivision(paddedWidth, max);
//    unsigned int gridh = ceilDivision(paddedHeight, max);
//
//    dim3 grids(gridw, gridh);
//    dim3 threads(max, max);
//
//
//    std::cout << "padW: " << paddedWidth << " padH: " << paddedHeight << std::endl;
//    std::cout << "gridw: " << gridw << " gridh: " << gridh  << " threads: " << max << std::endl;
//    // TODO multithread hipMalloc
//
//    uint8_t *orig;
//    uint8_t *eroded;
//    uint8_t *ker;
//
//    hipMalloc(&orig, sizeof (uint8_t) * (gridw * max) * (gridh * max));
//    hipMalloc(&eroded, sizeof (uint8_t) * imgWidth * imgHeight);
//    hipMalloc(&ker, sizeof (uint8_t) * kerSide * kerSide);
//
//    hipMemcpy(orig, paddedImg, sizeof (uint8_t) * (gridw * max) * (gridh * max), hipMemcpyHostToDevice);
//    hipMemcpy(ker, kernel, sizeof (uint8_t) * kerSide * kerSide, hipMemcpyHostToDevice);
//
//    printMat(paddedImg, 15, 15);
//    //printMat(result, 13, 13);
//
//    erosion<<<grids, threads>>>(orig, paddedWidth, paddedHeight, eroded, imgWidth, ker, kerSide);
//    //printf("orig: %d\n",  orig[indexY * width + indexX] );
//
//    hipDeviceSynchronize();
//
//    hipMemcpy(result, eroded, sizeof (uint8_t) * imgWidth * imgHeight, hipMemcpyDeviceToHost);
//
//    std::cout << static_cast<int>(result[7]) << std::endl;
//
//    hipDeviceSynchronize();
//    printMat(result, 13, 13);
//
//
//    hipFree(eroded);
//    hipFree(orig);
//    hipFree(ker);
//
//    return 0;
//
/*



    // data on HOST memory
    uint8_t *ma = new uint8_t[9] { 'E', 'A', 'j', '1', 'h', '0', 'q', 'd', 'w' };
    uint8_t *mo = new uint8_t[9] { 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x' };

    // GPU memory allocation
    uint8_t *mat;
    uint8_t *morphed;
    hipMalloc(&mat, 9 * sizeof (uint8_t));
    hipMalloc(&morphed, 9 * sizeof (uint8_t));

    // copy HOST memory to GPU memory
    hipMemcpy(mat, ma, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);
    hipMemcpy(morphed, mo, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);


    printMat(ma, 9);

    int nthreads = 9;
    const int n = 128 * 1024;
    int blockSize = 512;
    int nbBlocks = n / nthreads;
    dim3 grids(3, 3);


    // compute on GPU
    erosion<<<grids, threads>>>(mat, morphed, 3);

    // copy back GPU memory to HOST memory
    hipMemcpy(mo, morphed, sizeof (uint8_t) * 9, hipMemcpyDeviceToHost);

    printMat(mo, 9);

    return 0;

*/

}

