#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>
#include "../Image/Image.hh"
#include "../Chronometer.hh"
#include "../CPU/Morphology.hh"

__global__ void copy(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    printf("[x: %d, y: %d] ", indexX, indexY);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexY * width + indexX] = orig[indexY * width + indexX];
}

void printMat(uint8_t *mat, unsigned w, unsigned h) {

    std::cout << "[ " << std::endl;
for (int i = 0; i < w; ++i) {
        for (int j = 0; j < h; ++j) {
            std::cout << static_cast<unsigned>(mat[i * w + j]) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "]" << std::endl;
}




__global__ void erosion(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 255;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = min(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}

__global__ void dilation(uint8_t *orig, unsigned origWidth, unsigned origHeight,
                         uint8_t *morphed, unsigned width,
                         uint8_t *ker, unsigned side) {
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= origWidth - kerMid
        || indexY < kerMid || indexY >= origHeight - kerMid)
        return;

    morphed[(indexY - kerMid) * width + (indexX - kerMid)] = 0;
    
    for (int y = 0; y < side; ++y) {
        for (int x = 0; x < side; ++x) {
            if (ker[y * side + x]) {

                morphed[(indexX - kerMid) + (indexY - kerMid) * width] = max(
                        morphed[(indexY - kerMid) * width + (indexX - kerMid)],
                        orig[(indexX + x - kerMid) + (indexY + y - kerMid) * origWidth]
                        );
            }
        }

    }
}


int ceilDivision(int a, int b) {
    if (a % b == 0)
        return a / b;
    return a / b + 1;
}


Image benchDilate(Image &img, uint8_t *kernel, int iterations) {
    Chronometer chrono = Chronometer("[GPU] Dilate");

    Image padded = Image::addPadding(img, ker.size() / 2, 0);

    uint8_t *result = new uint8_t[img.getHeight() * img.getWidth()];

    unsigned int max = 512;
    unsigned int gridw = ceilDivision(padded.getWidth(), max);
    unsigned int gridh = ceilDivision(padded.getHeight(), max);

    dim3 grids(gridw, gridh);
    dim3 threads(max, max);


//    std::cout << "padW: " << paddedWidth << " padH: " << paddedHeight << std::endl;
//    std::cout << "gridw: " << gridw << " gridh: " << gridh  << " threads: " << max << std::endl;

    uint8_t *orig;
    uint8_t *morphed;
    uint8_t *ker;

    hipMalloc(&orig, sizeof (uint8_t) * (gridw * max) * (gridh * max));
    hipMalloc(&morphed, sizeof (uint8_t) * imgWidth * imgHeight);
    hipMalloc(&ker, sizeof (uint8_t) * kerSide * kerSide);

    hipMemcpy(orig, padded, sizeof (uint8_t) * (gridw * max) * (gridh * max), hipMemcpyHostToDevice);
    hipMemcpy(ker, kernel, sizeof (uint8_t) * kerSide * kerSide, hipMemcpyHostToDevice);

    printMat(padded, 15, 15);
    //printMat(result, 13, 13);

    erosion<<<grids, threads>>>(orig, padded.getWidth(), padded.getHeight(), morphed, imgWidth, ker, kerSide);
    //printf("orig: %d\n",  orig[indexY * width + indexX] );

    hipDeviceSynchronize();

    hipMemcpy(result, morphed, sizeof (uint8_t) * imgWidth * imgHeight, hipMemcpyDeviceToHost);

    std::cout << static_cast<int>(result[7]) << std::endl;

    hipDeviceSynchronize();
//    printMat(result, 13, 13);

    hipFree(morphed);
    hipFree(orig);
    hipFree(ker);

    return Image(img.getWidth(), img.getHeight(), result);
}

int main(int argc, char **argv)
{

    Image img = Image::fromPPM("../Data/RealSnake.ppm", Image::ImportType::BINARY);
    unsigned char *kernel = Morphology::kerSquareArray(3);
    benchDilate(img, kernel, 1);

    return 0;
}

