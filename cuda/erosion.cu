#include <hip/hip_runtime.h>
//#include <helper_cuda.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>

__global__ void copy(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    printf("[x: %d, y: %d] ", indexX, indexY);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexY * width + indexX] = orig[indexY * width + indexX];
}

void printMat(uint8_t *mat, unsigned w, unsigned h) {

    std::cout << "[ " << std::endl;
for (int i = 0; i < w; ++i) {
        for (int j = 0; j < h; ++j) {
            std::cout << static_cast<unsigned>(mat[i * w + j]) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "]" << std::endl;
}




__global__ void erosion(uint8_t *orig, uint8_t *morphed, unsigned width, unsigned height, unsigned side) {
    printf("boop ");
    int kerMid = side / 2;
    int indexX = blockIdx.x * blockDim.x + threadIdx.x + kerMid;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y + kerMid;

    if (indexX < kerMid || indexX >= width - kerMid
        || indexY < kerMid || indexY >= height - kerMid)
        return;

    morphed[indexY * width + indexX] = 255;

    return;
    for (int y = -kerMid; y <= kerMid; ++y) {
        for (int x = -kerMid; x <= kerMid; ++x) {
            morphed[indexX + indexY * width] = min(
                    morphed[indexY * width + indexX],
                    orig[(indexX + x) + (indexY + y) * width]
                    );
        }

    }
}


int ceilDivision(int a, int b) {
    if (a % b == 0)
        return a / b;
    return a / b + 1;
}


int main(int argc, char **argv)
{

    // img
    uint8_t img[9] = { 1, 1, 1,
                       1, 1, 1,
                       1, 1, 1 };
    unsigned int imgWidth = 3;
    unsigned int imgHeight = 3;
    // kernel
    uint8_t kernel[9] = { 1, 1, 1,
                          1, 1, 1,
                          1, 1, 1 };
    unsigned int kerSide = 3;
    // paddedImg
    uint8_t paddedImg[25] =   { 0, 0, 0, 0, 0,
                                0, 0, 0, 0, 0,
                                0, 0, 1, 0, 0,
                                0, 0, 0, 0, 0,
                                0, 0, 0, 0, 0 };

    uint8_t *result = new uint8_t[25]();


    unsigned int paddedWidth  = imgWidth  + (kerSide  / 2) * 2;
    unsigned int paddedHeight = imgHeight + (kerSide / 2) * 2;

    unsigned int max = 8;
    dim3 grids(ceilDivision(paddedWidth, max), ceilDivision(paddedHeight, max));
    dim3 threads(max, max);


    std::cout << "padW: " << paddedWidth << " padH: " << paddedHeight << std::endl;
    std::cout << "grids: " << ceilDivision(paddedWidth, max) << " threads: " << max << std::endl;
    // TODO multithread cudaMalloc
    
    uint8_t *orig;
    uint8_t *eroded;
    uint8_t *ker;

    hipMalloc(&orig, sizeof (uint8_t) * paddedWidth * paddedHeight);
    hipMalloc(&eroded, sizeof (uint8_t) * paddedWidth * paddedHeight);
    hipMalloc(&ker, sizeof (uint8_t) * kerSide * kerSide);

    hipMemcpy(orig, paddedImg, sizeof (uint8_t) * paddedWidth * paddedHeight, hipMemcpyHostToDevice);
    hipMemcpy(ker, kernel, sizeof (uint8_t) * kerSide * kerSide, hipMemcpyHostToDevice);

    printMat(paddedImg, 5, 5);

    erosion<<<grids, threads>>>(orig, eroded, paddedWidth, paddedHeight, kerSide);

    hipMemcpy(eroded, result, sizeof (uint8_t) * paddedWidth * paddedHeight, hipMemcpyDeviceToHost);

    printMat(result, 5, 5);


    return 0;

/*



    // data on HOST memory
    uint8_t *ma = new uint8_t[9] { 'E', 'A', 'j', '1', 'h', '0', 'q', 'd', 'w' };
    uint8_t *mo = new uint8_t[9] { 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x' };

    // GPU memory allocation
    uint8_t *mat;
    uint8_t *morphed;
    cudaMalloc(&mat, 9 * sizeof (uint8_t));
    cudaMalloc(&morphed, 9 * sizeof (uint8_t));

    // copy HOST memory to GPU memory
    cudaMemcpy(mat, ma, sizeof(uint8_t) * 9, cudaMemcpyHostToDevice);
    cudaMemcpy(morphed, mo, sizeof(uint8_t) * 9, cudaMemcpyHostToDevice);


    printMat(ma, 9);

    int nthreads = 9;
    const int n = 128 * 1024;
    int blockSize = 512;
    int nbBlocks = n / nthreads;
    dim3 grids(3, 3);


    // compute on GPU
    erosion<<<grids, threads>>>(mat, morphed, 3);

    // copy back GPU memory to HOST memory
    cudaMemcpy(mo, morphed, sizeof (uint8_t) * 9, cudaMemcpyDeviceToHost);

    printMat(mo, 9);

    return 0;

*/

}

