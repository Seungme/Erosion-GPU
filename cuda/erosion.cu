#include <hip/hip_runtime.h>
//#include <helper_cuda.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>

__global__ void helloOnGPU()
{
    printf("Hello World on GPU!\n");
}

__global__ void erosion(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    // int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    //printf("%d ", indexX);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexX] = orig[indexX];
}

void printMat(uint8_t *mat, unsigned size) {

    std::cout << "[ ";
    for (int i = 0; i < size; ++i) {
        std::cout << mat[i] << " ";
    }
    std::cout << "]" << std::endl;
}

int main(int argc, char **argv)
{

    // helloOnGPU<<<1, 10>>>();

    uint8_t *ma = new uint8_t[9] { 'E', 'A', 'j', '1', 'h', '0', 'q', 'd', 'w' };
    uint8_t *mo = new uint8_t[9] { 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x' };

    uint8_t *mat;
    uint8_t *morphed;
    hipMalloc(&mat, 9 * sizeof (uint8_t));
    hipMalloc(&morphed, 9 * sizeof (uint8_t));

    hipMemcpy(mat, ma, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);
    hipMemcpy(morphed, mo, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);


    printMat(ma, 9);

    int nthreads = 9;
    const int n = 128 * 1024;
    int blockSize = 512;
    int nbBlocks = n / nthreads;
    // dim3 grids(3, 3);
    // dim3 threads
    // std::cout << "nbBlocks: " << nbBlocks << "  blockSize: " << blockSize << std::endl;
    
    erosion<<<1, 9>>>(mat, morphed, 3);

    hipMemcpy(mo, morphed, sizeof (uint8_t) * 9, hipMemcpyDeviceToHost);

    printMat(mo, 9);

    return 0;
}

