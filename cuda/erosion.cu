#include <hip/hip_runtime.h>
//#include <helper_cuda.h>
#include <cstdint>
#include <iostream>
#include <stdio.h>

__global__ void erosion(uint8_t *orig, uint8_t *morphed, unsigned width) {
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d, %c ", indexX, orig[indexX]);
    printf("[x: %d, y: %d] ", indexX, indexY);
    // printf("%c ", orig[indexX]);
    // morphed[indexY * width + indexX] = orig[indexY * width + indexX];

    morphed[indexY * width + indexX] = orig[indexY * width + indexX];
}

void printMat(uint8_t *mat, unsigned size) {

    std::cout << "[ ";
    for (int i = 0; i < size; ++i) {
        std::cout << mat[i] << " ";
    }
    std::cout << "]" << std::endl;
}

int main(int argc, char **argv)
{

    // data on HOST memory
    uint8_t *ma = new uint8_t[9] { 'E', 'A', 'j', '1', 'h', '0', 'q', 'd', 'w' };
    uint8_t *mo = new uint8_t[9] { 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x', 'x' };

    // GPU memory allocation
    uint8_t *mat;
    uint8_t *morphed;
    hipMalloc(&mat, 9 * sizeof (uint8_t));
    hipMalloc(&morphed, 9 * sizeof (uint8_t));

    // copy HOST memory to GPU memory
    hipMemcpy(mat, ma, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);
    hipMemcpy(morphed, mo, sizeof(uint8_t) * 9, hipMemcpyHostToDevice);


    printMat(ma, 9);

    int nthreads = 9;
    const int n = 128 * 1024;
    int blockSize = 512;
    int nbBlocks = n / nthreads;
    dim3 grids(3, 3);
    dim3 threads(3, 3);

    // compute on GPU
    erosion<<<grids, 1>>>(mat, morphed, 3);

    // copy back GPU memory to HOST memory
    hipMemcpy(mo, morphed, sizeof (uint8_t) * 9, hipMemcpyDeviceToHost);

    printMat(mo, 9);

    return 0;
}

